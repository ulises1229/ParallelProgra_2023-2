#include "hip/hip_runtime.h"
// Author: Ulises Olivares
// uolivares@unam.mx
// Oct 1,2020

#include <iostream>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <vector>

#define N 90000000

using namespace std;

void generateRandom(int *h_a);
void parallelAddition();
void serialAddition();
void compareVectors(int *parallelC, int *serialC);

// Variables globales
double serialTimer = 0;
double parallelTimer = 0;

int *h_a, *h_b, *h_c, *serialC;
int *d_a, *d_b, *d_c;
int size = N * sizeof(int);


// Kernel vectorAdd
__global__ void vectorAdd(int *h_a, int *h_b, int *h_c){
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	h_c[index] = h_a[index] + h_b[index];
}

int main(){
	h_a = (int *) malloc(size);
	h_b = (int *) malloc(size);
	h_c = (int *) malloc(size);
	serialC = (int *) malloc(size);


	// initialize arrays with random numbers
	generateRandom(h_a);
	generateRandom(h_b);


	parallelAddition();

	serialAddition();

	compareVectors(h_c, serialC);

	cout << "Speed-up: " << serialTimer / (parallelTimer /1000)<< "X"<<endl; 

	free(h_a); free(h_b); free(h_c); free(serialC);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;
}

void parallelAddition(){

	// Reservar memoria en device
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// Transferir datos de host h_a device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);


	int blocks = ceil(N / 1024) + 1;
	int threads = 1024;

	// define timers 
	hipEvent_t start, stop;

	// events to take time
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);

	// Launch kernel
	vectorAdd<<<blocks, threads>>>(d_a, d_b, d_c);

	hipEventRecord(stop,0);

	hipEventSynchronize(stop);

	float parallelTimer = 0;

	hipEventElapsedTime(&parallelTimer, start, stop);

	cout<< "Elapsed parallel timer: " << parallelTimer << " ms, " << parallelTimer / 1000 << " secs" <<endl;

	hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
}

void compareVectors(int *parallelC, int *serialC){
	
	int diff = 0;
	for(int i= 0; i<N; i++)
		if(parallelC[i] != serialC[i]){
			diff++;
			cout << "Parallel: " << parallelC[i] << " Serial: " << serialC[i] <<endl; 
		}

	
	if(diff>0){
		cout<< diff <<" elements different" << endl;

	}
	else
		cout << "Vectors are equal!..." << endl;
}

void serialAddition(){
	
	clock_t start = clock();
	for(int i= 0; i<N; i++)
		h_c[i] = h_a[i] + h_b[i];
	clock_t end = clock();

	serialTimer = double (end-start) / double(CLOCKS_PER_SEC);
	cout << "Elapsed time serial: " << serialTimer << endl;
}


void generateRandom(int *h_a){
	
	// Initialize seed
	srand(time(NULL));

	for(int i=0; i<N; i++){
		h_a[i] = rand() % 100 +1;
	}

}
