
void generateLinearSystem(unsigned char *A, unsigned char* B, unsigned char* x, int rowCount, int columnCount, int percent)
{

	unsigned long int row, col;
	int dice;

	percent = (percent*64)/100;

	for(row = 0; row < rowCount; row++)
	{
		for(col = 0; col < columnCount; col++)
		{
			dice = (((rand() >> 8) & 63)+1);
			if(dice <= percent)
				A[row*columnCount+col] = (unsigned char) 1;
			else
				A[row*columnCount+col] = (unsigned char) 0;
		}
	}

	for(col = 0; col < columnCount; col++)
	{
		x[col] = (unsigned char)((rand() >> 8) % 2);
	}

	for(row = 0; row < rowCount; row++)
	{
		B[row] = 0;
		for(col = 0; col < columnCount; col++)
		{
			B[row] ^= (A[row*columnCount+col] & x[col]);
		}
	}

	return;
}

void writeVectorToFile(const char* filename, unsigned char* vector, int length)
{
	FILE *fp;
	int col;

	fp = fopen(filename, "w");

	if(fp == NULL)
	{
		printf("Failed to open file '%s' for writing.\n", filename);
		return;
	}

	for(col = 0; col < length; col++)
	{

		fprintf(fp, "%u\n", (unsigned int) vector[col]);
	}

	fclose(fp);
}

void printMatrix(unsigned char* matrixA, int rowCount, int columnCount)
{
	int row, col;
	printf("Number of columns = %d\n", columnCount);
	printf("Number of rows = %d\n", rowCount);

	printf("      | ");
	for(col = 0; col < columnCount; col++)
	{
		printf("%d", col % 10);
	}
	printf("\n");

	printf("--------");
	for(col = 0; col < columnCount; col++)
	{
		printf("-");
	}
	printf("\n");

	for(row = 0; row < rowCount; row++)
	{
		printf("%5d | ", row);
		for(col = 0; col < columnCount; col++)
		{
			printf("%u", (unsigned int)matrixA[row*columnCount + col]);
		}
		printf("\n");
	}
}

void packLinearSystem(unsigned int* packedAB, unsigned char* A, unsigned char* B, int rowCount, int columnCount)
{
	int i, j, k;
	int packedColumnCount = intCeilDiv((columnCount + 1), PACK_SIZE); // One extra column for B
	unsigned int temp;

	long int index, packIndex, bitIndex;

	for(i = 0; i < rowCount; i++)
	{
		for(j = 0; j < packedColumnCount; j++)
		{
			temp = 0;
			for(k = 0; k < PACK_SIZE; k++)
			{
				if((j*PACK_SIZE + k) < columnCount) 
				{
					index = (long int)i * columnCount + (j*PACK_SIZE + k);
					if(A[index])
					{
						temp |= (1 << k);
					}
				}

			}
			index = (long int)i * packedColumnCount + j;
			packedAB[index] = temp;
		}
	}

	for(i = 0; i < rowCount; i++)
	{
		index = i * packedColumnCount + (packedColumnCount - 1); // B is embedded as last column
		bitIndex = columnCount % PACK_SIZE;
		if(B[i])
		{
			packedAB[index] |= (1 << bitIndex);
		}
	}
}

void unpackLinearSystem(unsigned char* A, unsigned char* B, unsigned int* packedAB, int rowCount, int columnCount)
{
	int i, j, k;
	int packedColumnCount = intCeilDiv((columnCount + 1), PACK_SIZE); // One extra column for B
	unsigned int temp;
	long int index, packIndex, bitIndex;

	for(i = 0; i < rowCount; i++)
	{
		for(j = 0; j < packedColumnCount; j++)
		{
			index = (long int)i * packedColumnCount + j;
			temp = packedAB[index];
			for(k = 0; k < PACK_SIZE; k++)
			{
				if(j*PACK_SIZE + k < columnCount)
				{
					index = (long int)i * columnCount + j*PACK_SIZE + k;
					A[index] = (temp >> k) & 1;
				}
			}
		}
	}
	for(i = 0; i < rowCount; i++)
	{
		index = i * packedColumnCount  + packedColumnCount - 1; // B is embedded as last column
		bitIndex = columnCount % PACK_SIZE;
		B[i] = (unsigned char) ((packedAB[index] >> bitIndex) & 1);

	}
}

void transposeMatrixCPU(unsigned int *transposeA, unsigned int *A, int  rowCount, int columnCount)
{
	int row, col;
	long int index1, index2;
	for(row = 0; row < rowCount; row++)
	{
		for(col = 0; col < columnCount; col++)
		{
			index1 = (long int)col*rowCount + row;
			index2 = (long int)row*columnCount + col;
			transposeA[index1] = A[index2];
		}
	}
}

void readLinearSystemFromFile(const char* filename, unsigned char* matrixA, unsigned char* B, int rowCount, int columnCount)
{
	int row, col;
	FILE* fp;
	unsigned int temp;
	long int index;

	fp = fopen(filename, "r");
	for(row = 0; row < rowCount; row++)
	{
		for(col = 0; col < columnCount; col++)
		{
			fscanf(fp, "%u, ", &temp);
			index = (long int)row*(columnCount) + col;
			matrixA[index] = (unsigned char) temp;
		}
		fscanf(fp, "%u\n", &temp);
		B[row] = temp;
	}
	fclose(fp);
}

void writeLinearSystemFromFile(const char* filename, unsigned char* matrixA, unsigned char* B, int rowCount, int columnCount)
{
	int row, col;
	FILE* fp;
	long int index;

	fp = fopen(filename, "w");

	if(fp == NULL)
	{
		printf("Failed to open file %s for writing.\n", filename);
		return;
	}
	for(row = 0; row < rowCount; row++)
	{
		for(col = 0; col < columnCount; col++)
		{
			index = (long int)row*columnCount + col;
			fprintf(fp, "%u, ", (unsigned int)matrixA[index]);
		}
		fprintf(fp, "%u\n", B[row]);
	}
	fclose(fp);
}

int backsubstitution(unsigned char* matrixA, unsigned char* B, unsigned char* x, int number_of_equations, int number_of_variables)
{
	int row, col, var;
	int* pivoteRowIndex;
	long int index;

	memset(x, 0, sizeof(unsigned char) * number_of_variables);
	pivoteRowIndex = (int*) malloc(sizeof(int)*number_of_equations);

	for(row = 0; row < number_of_equations; row++)
	{
		pivoteRowIndex[row] = -1;
	}

	for(row = 0; row < number_of_equations; row++)
	{
		for(col = 0; col < number_of_variables; col++)
		{
			index = (long int)row*number_of_variables+col;
			if(matrixA[index] != 0)
				break;
		}
		if(col != number_of_variables)
		{
			// printf("Row %d is pivot for col %d\n", row, col);
			pivoteRowIndex[col] = row;
		}
		else
		{
			// printf("Row %d is NULL\n", row);
		}
	}

	for(var = number_of_variables - 1; var >= 0; var--)
	{
		if(pivoteRowIndex[var] == -1)
		{
			printf("backsubstitution failed at %d\n", var);
			break;
		}
		row = pivoteRowIndex[var];
		x[var] = B[row];
		for(col = var+1; col < number_of_variables; col++)
		{
			index = (long int)row*number_of_variables+col;
			x[var] ^= (matrixA[index] & x[col]);
		}
	}

	free(pivoteRowIndex);
	return 0;
}
