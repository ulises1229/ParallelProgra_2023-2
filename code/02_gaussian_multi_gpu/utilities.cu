
void _report(hipError_t result, char const *const func, const char *const file, int const line)
{
	if(result)
	{
		fprintf(stderr, "CUDA error at %s:%d code = %d (%s) \"%s\" \n",
				file, line, result, hipGetErrorString(result), func);
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(-1);
	}
}

__device__ unsigned int bfe(unsigned int x, unsigned int bit, unsigned int numBits) 
{
	unsigned int ret;
	asm("bfe.u32 %0, %1, %2, %3;" : "=r"(ret) : "r"(x), "r"(bit), "r"(numBits));
	return ret;
}

int setupPeerToPeer(int GPUCount)
{

	int canAccessPeer;
	for(int i = 0; i < GPUCount; i++)
	{
		checkCudaErrors( hipSetDevice(i) );
	}

	for(int i = 0; i < GPUCount; i++)
	{
		checkCudaErrors( hipSetDevice(i) );
		for(int j = 0; j < GPUCount; j++)
		{
			if(i == j) continue;
			checkCudaErrors( hipDeviceCanAccessPeer(&canAccessPeer, i, j) );
			if(canAccessPeer)
			{
				printf("Can access memory of device %d from device %d\n", j, i);
				checkCudaErrors( hipDeviceEnablePeerAccess(j, 0) );
			}    
			else
			{
				printf("Can not access memory of device %d from device %d\n", j, i);
				return 0;
			}

		}
	}
	return 1;
}

int testPeerToPeer(int GPUCount)
{
	char** buffer;

	int buffersize = 1024 * sizeof(char);
	buffer = (char**) malloc(GPUCount * sizeof(char*));

	for(int i = 0; i < GPUCount; i++)
	{
		checkCudaErrors( hipSetDevice(i) );
		checkCudaErrors( hipMalloc((void**)&buffer[i], buffersize) );
	}

	for(int i = 0; i < GPUCount; i++)
	{
		for(int j = 0; j < GPUCount; j++)
		{
			if(i == j) continue;
			checkCudaErrors( hipMemcpyPeer(buffer[i], i, buffer[j], j, buffersize) );
		}
	}

	for(int i = 0; i < GPUCount; i++)
	{
		checkCudaErrors( hipFree(buffer[i]) );
	}
	return 1;
}
